#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include "engine.h"

// Kernel to initialize CURAND state for each thread.
__global__ void init_particles_kernel(hiprandState *states, unsigned long seed, ParticleSystem ps, int num_particles, float max_speed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_particles)
    {
        // hiprand_init(seed, sequence number, offset, state pointer)
        hiprand_init(seed, idx, 0, &states[idx]);
        ps.x[idx] = hiprand_uniform(&states[idx]);
        ps.y[idx] = hiprand_uniform(&states[idx]);
        ps.vx[idx] = max_speed * hiprand_uniform(&states[idx]);
        ps.vy[idx] = max_speed * hiprand_uniform(&states[idx]);
        ps.mass[idx] = 1.0f;
    }
}

__global__ void update_particles_kernel(ParticleSystem ps_new, ParticleSystem ps_old, int num_particles, float dt, float G, float epsilon)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_particles)
    {
        float net_accelx = 0.0f;
        float net_accely = 0.0f;
        for (int j = 0; j < num_particles; j++)
        {
            if (idx == j)
            {
                continue;
            }
            float dx = ps_old.x[idx] - ps_old.x[j];
            float dy = ps_old.y[idx] - ps_old.y[j];
            float dist_sq = dx * dx + dy * dy + epsilon;  // avoid division by 0
            float accel = -G * ps_old.mass[j] / dist_sq;
            net_accelx += dx * accel;
            net_accely += dy * accel;
        }
        ps_new.vx[idx] = ps_old.vx[idx] + net_accelx * dt;
        ps_new.vy[idx] = ps_old.vy[idx] + net_accely * dt;
        ps_new.x[idx] = ps_old.x[idx] + ps_new.vx[idx] * dt;
        ps_new.y[idx] = ps_old.y[idx] + ps_new.vy[idx] * dt;

        // Toroidal boundaries: Wrap positions around the domain [0,1]
        if (ps_new.x[idx] < 0.0f)
            ps_new.x[idx] += 1.0f;
        else if (ps_new.x[idx] >= 1.0f)
            ps_new.x[idx] -= 1.0f;
        if (ps_new.y[idx] < 0.0f)
            ps_new.y[idx] += 1.0f;
        else if (ps_new.y[idx] >= 1.0f)
            ps_new.y[idx] -= 1.0f;
    }
}

void allocate_curand_states(hiprandState **states, int num_particles)
{
    size_t size = num_particles * sizeof(hiprandState);
    hipMalloc(states, size);
}

void free_curand_states(hiprandState *states)
{
    hipFree(states);
}

extern "C" void allocate_particle_system(ParticleSystem *ps, int num_particles)
{
    size_t size = num_particles * sizeof(float);

    hipMalloc(&ps->x, size);
    hipMalloc(&ps->y, size);
    hipMalloc(&ps->vx, size);
    hipMalloc(&ps->vy, size);
    hipMalloc(&ps->mass, size);
}

extern "C" void free_particle_system(ParticleSystem *ps)
{
    hipFree(ps->x);
    hipFree(ps->y);
    hipFree(ps->vx);
    hipFree(ps->vy);
    hipFree(ps->mass);
}

extern "C" void initialize_simulation(ParticleSystem *ps_start, int num_particles, float max_speed)
{
    // Allocate hiprand state
    hiprandState *d_states;
    allocate_curand_states(&d_states, num_particles);

    dim3 threadsPerBlock(256);
    dim3 numBlocks((num_particles + threadsPerBlock.x - 1) / threadsPerBlock.x);

    //
    init_particles_kernel<<<numBlocks, threadsPerBlock>>>(d_states, time(NULL), *ps_start, num_particles, max_speed);

    hipDeviceSynchronize();

    free_curand_states(d_states);
}

extern "C" void launch_update_particles(ParticleSystem *ps_new, ParticleSystem *ps_old, int num_particles, float dt, float G, float epsilon)
{
    dim3 threadsPerBlock(256);
    dim3 numBlocks((num_particles + threadsPerBlock.x - 1) / threadsPerBlock.x);
    update_particles_kernel<<<numBlocks, threadsPerBlock>>>(*ps_new, *ps_old, num_particles, dt, G, epsilon);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA error: %s", hipGetErrorString(err));
    }
}

extern "C" void swap_particle_systems(ParticleSystem *ps1, ParticleSystem *ps2)
{
    ParticleSystem temp = *ps1;
    *ps1 = *ps2;
    *ps2 = temp;
}

extern "C" void copy_ps_dev_to_host(ParticleSystem *host, ParticleSystem *dev, int num_particles)
{
    size_t size = num_particles * sizeof(float);

    hipMemcpy(host->x, dev->x, size, hipMemcpyDeviceToHost);
    hipMemcpy(host->y, dev->y, size, hipMemcpyDeviceToHost);
    hipMemcpy(host->vx, dev->vx, size, hipMemcpyDeviceToHost);
    hipMemcpy(host->vy, dev->vy, size, hipMemcpyDeviceToHost);
    hipMemcpy(host->mass, dev->mass, size, hipMemcpyDeviceToHost);
}

extern "C" void copy_ps_dev_to_dev(ParticleSystem *dev_new, ParticleSystem *dev_old, int num_particles)
{
    size_t size = num_particles * sizeof(float);

    hipMemcpy(dev_new->x, dev_old->x, size, hipMemcpyDeviceToDevice);
    hipMemcpy(dev_new->y, dev_old->y, size, hipMemcpyDeviceToDevice);
    hipMemcpy(dev_new->vx, dev_old->vx, size, hipMemcpyDeviceToDevice);
    hipMemcpy(dev_new->vy, dev_old->vy, size, hipMemcpyDeviceToDevice);
    hipMemcpy(dev_new->mass, dev_old->mass, size, hipMemcpyDeviceToDevice);
}